#include "hip/hip_runtime.h"
#include <iostream>
#include <gtest/gtest.h>
#include "util.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

TEST(tensor_transformation, reshape_input_tensor) {
    // Set filter size
    int filter_h = 2;
    int filter_w = 2;

    // Create an input tensor [C=1, H=3, W=3] with values from 1 to 9
    torch::Tensor input = torch::arange(1, 1 * 3 * 3 + 1).view({1, 3, 3}).to(torch::kFloat);

    // Extract input dimensions
    int C = input.size(0);  // Number of channels
    int H = input.size(1);  // Height
    int W = input.size(2);  // Width

    // Call the tensor_transformation function
    torch::Tensor output = tensor_transformation(input, filter_h, filter_w);

    // Expected output dimensions
    int H_new = H - filter_h + 1;  
    int W_new = W - filter_w + 1;
    std::vector<int64_t> expected_shape = {H_new, W_new, filter_h, filter_w, C};

    // Test the shape of the output tensor
    EXPECT_EQ(output.sizes().vec(), expected_shape) << "Output shape mismatch.";

    // Expected output tensor values (manually computed patches)
    torch::Tensor expected_output = torch::tensor(
        {
            {{{{1, 2}, {4, 5}}, {{2, 3}, {5, 6}}},  // Patches in (hi, wi)
             {{{4, 5}, {7, 8}}, {{5, 6}, {8, 9}}}},
        },
        torch::kFloat
    );

    // Flatten both tensors for comparison
    torch::Tensor flattened_output = output.flatten();
    torch::Tensor flattened_expected = expected_output.flatten();

    EXPECT_EQ(flattened_output.sizes(), flattened_expected.sizes())
        << "Flattened shape mismatch.";

    // Assert values
    EXPECT_TRUE(torch::allclose(flattened_output, flattened_expected))
        << "Flattened tensor values mismatch.\nExpected:\n"
        << flattened_expected << "\nActual:\n"
        << flattened_output;
}

torch::Tensor create_diagonal_matrix(int rows, int cols) {
        torch::Tensor diag = torch::eye(std::min(rows, cols)).to(torch::kFloat);
        if (rows > cols) {
            return torch::cat({diag, torch::zeros({rows - cols, cols}).to(torch::kFloat)}, 0);
        } else if (cols > rows) {
            return torch::cat({diag, torch::zeros({rows, cols - rows}).to(torch::kFloat)}, 1);
        }
        return diag;
};

TEST(tensor_contraction_sgemv, einsum){
    torch::Tensor test = torch::arange(1, 2*3*4*5 + 1).view({2, 3, 4, 5}).to(torch::kFloat).contiguous();

    std::vector<torch::Tensor> factors = {
        create_diagonal_matrix(7, 6),
        create_diagonal_matrix(3, 6),
        create_diagonal_matrix(4, 6),
        create_diagonal_matrix(5, 1)
        // torch::tensor({1, 0, 0, 0, 0, 0, 1, 0, 0, 0}).reshape({5, 2}).to(torch::kFloat)
        // torch::tensor({1, 0, 0, 0, 0}).to(torch::kFloat)
        // torch::tensor({1, 0, 0, 0, 0}).reshape({5, 1}).to(torch::kFloat)
    };

    // Assign tensor sizes to a variable
    std::vector<int64_t> tensor_sizes = test.sizes().vec();
    int batch_count = factors[3].size(1); // factors[3].size(0)/tensor_sizes[3];
    int three_size = 1;
    for(int i=0; i<3; i++){
        three_size *= tensor_sizes[i];
    }

    torch::Tensor y_3 = torch::zeros({three_size}).to(torch::kFloat); // y initialized with 0

    float *d_A, *d_x3, *d_y3;
    // Allocate device memory
    hipMalloc((void**)&d_A, three_size * tensor_sizes[3] * sizeof(float));
    hipMalloc((void**)&d_x3, tensor_sizes[3] * batch_count * sizeof(float));
    hipMalloc((void**)&d_y3, (three_size) * sizeof(float));

    // Copy data to device memory
    hipMemcpy(d_A, test.data_ptr<float>(), (three_size) * tensor_sizes[3] * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x3, factors[3].data_ptr<float>(), tensor_sizes[3] * batch_count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y3, y_3.data_ptr<float>(), (three_size) * sizeof(float), hipMemcpyHostToDevice);
    // Initialize cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Set parameters for SGEMV
    float alpha = 1.;
    float beta = 1;
    hipblasOperation_t trans = HIPBLAS_OP_T;
    int lda = tensor_sizes[3];             // Leading dimension of A (number of rows)
    int stride_A = 0;                 // Reuse the same d_A for all batches
    int stride_x = tensor_sizes[3];   // Stride between columns in factors[3]
    int stride_y = 0;        // No stride for d_y3, accumulate into a single vector

    hipblasSgemv(
        handle, trans, tensor_sizes[3], three_size,
        &alpha, d_A, lda,
        d_x3, 1,
        &beta, d_y3, 1
    );

    // Allocate host memory to retrieve the data
    float* h_y = new float[(three_size) * batch_count];
    hipMemcpy(h_y, d_y3, (three_size) * batch_count * sizeof(float), hipMemcpyDeviceToHost);

    torch::Tensor reshaped = test.view({24, 5}); // Reshape into (24, 5)
    torch::Tensor transposed = reshaped.t(); // Transpose to (5, 24)
    torch::Tensor expected = transposed.slice(0, 0, 1).squeeze(); // Extract first column

    // Perform the comparison
    for (int i = 0; i < expected.size(0); ++i) {
        EXPECT_EQ(h_y[i], expected[i].item<float>()) << "Mismatch at index " << i;
    }

    // Clean up memory
    delete[] h_y;
    hipFree(d_A);
    hipFree(d_x3);
    hipFree(d_y3);
    hipblasDestroy(handle);
}

TEST(tensor_contraction_sgemm, einsum){
    torch::Tensor test = torch::arange(1, 2*3*4*5 + 1).view({2, 3, 4, 5}).to(torch::kFloat).contiguous();

    std::vector<torch::Tensor> factors = {
        create_diagonal_matrix(7, 6),
        create_diagonal_matrix(3, 6),
        create_diagonal_matrix(4, 6),
        create_diagonal_matrix(5, 6),
    };

    // Assign tensor sizes to a variable
    std::vector<int64_t> tensor_sizes = test.sizes().vec();
    int batch_count = factors[3].size(1);
    int three_size = 1;
    for(int i=0; i<3; i++){
        three_size *= tensor_sizes[i];
    }

    torch::Tensor y_3 = torch::zeros({three_size * batch_count}).to(torch::kFloat); // y initialized with 0
    torch::Tensor ones = torch::ones({batch_count}).to(torch::kFloat); // one initialized with 1
    // torch::Tensor y_2 = torch::zeros({tensor_sizes[0]*tensor_sizes[1]}).to(torch::kFloat); // y initialized with 0
    // torch::Tensor y_1 = torch::zeros({tensor_sizes[0]}).to(torch::kFloat); // y initialized with 0
    // Device pointers
    // float *d_A, *d_x1, *d_x2, *d_x3, *d_y;
    float *d_A, *d_x3, *d_y3, *d_row_sums, *d_ones;
    // Allocate device memory
    hipMalloc((void**)&d_A, three_size * tensor_sizes[3] * sizeof(float));
    hipMalloc((void**)&d_x3, tensor_sizes[3] * batch_count * sizeof(float));
    // hipMalloc((void**)&d_x2, sizeof(factors[2]));//factors[2]
    // hipMalloc((void**)&d_x1, sizeof(factors[1]));//factors[1]
    // hipMalloc((void**)&d_yy, sizeof(factors[0]));//factors[0]
    hipMalloc((void**)&d_y3, (three_size) * batch_count * sizeof(float));
    hipMalloc((void**)&d_ones, batch_count * sizeof(float));
    hipMalloc((void**)&d_row_sums, (three_size) * sizeof(float));
    // hipMalloc((void**)&d_y2, sizeof(y_2));
    // hipMalloc((void**)&d_y1, sizeof(y_1));

    // Copy data to device memory
    hipMemcpy(d_A, test.data_ptr<float>(), (three_size) * tensor_sizes[3] * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x3, factors[3].data_ptr<float>(), tensor_sizes[3] * batch_count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y3, y_3.data_ptr<float>(), (three_size) * batch_count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ones, ones.data_ptr<float>(), batch_count * sizeof(float), hipMemcpyHostToDevice);
    // Initialize cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Set parameters for SGEMV
    float alpha = 1.;
    float beta = 1;
    hipblasOperation_t trans = HIPBLAS_OP_T;
    int lda = tensor_sizes[3];             // Leading dimension of A (number of rows)

    hipblasSgemm(
        handle,
        trans, trans,
        three_size, batch_count, tensor_sizes[3],
        &alpha,
        d_A, tensor_sizes[3],//(5x24)^T
        d_x3, batch_count,//(6x5)^T
        &beta,
        d_y3, three_size//(24x6) column-major
    );

    // Allocate host memory to retrieve the data
    float* h_y = new float[(three_size) * batch_count];
    hipMemcpy(h_y, d_y3, (three_size) * batch_count * sizeof(float), hipMemcpyDeviceToHost);

    // // Print the result vector y
    // std::cout << "Result vector (y) first column:" << std::endl;
    // for (int i = 0; i < (three_size) * batch_count; i++) {
    //     std::cout << h_y[i] << std::endl;
    // }
    torch::Tensor reshaped = test.view({three_size, tensor_sizes[3]}); // Reshape into (24, 5)
    torch::Tensor transposed = reshaped.t(); // Transpose to (5, 24)
    torch::Tensor flattened = transposed.contiguous().flatten();
    // Perform the comparison
    int min = (batch_count < tensor_sizes[3]) ? batch_count : tensor_sizes[3];
    for (int i = 0; i < (three_size) * min; ++i) {
        EXPECT_EQ(h_y[i], flattened[i].item<float>()) << "Mismatch at index " << i;
    }

    hipblasSgemv(
        handle,
        HIPBLAS_OP_N,            // No transpose for d_y3
        three_size, batch_count, // Rows = 24, Columns = 6
        &alpha,                 // Scaling factor (1.0)
        d_y3, three_size,       // Matrix d_y3
        d_ones, 1,              // Vector d_ones
        &beta,                  // Scaling factor for existing d_row_sums (0.0)
        d_row_sums, 1           // Output: row sums
    );

    // Allocate host memory to retrieve the data
    float* h_row_sums = new float[three_size];
    hipMemcpy(h_row_sums, d_row_sums, three_size * sizeof(float), hipMemcpyDeviceToHost);
    torch::Tensor result = reshaped.sum(1); // Result is a (24,) tensor
    auto expected_row_sums = result.data_ptr<float>(); // Access raw data
     for (int i = 0; i < three_size; i++) {
        EXPECT_EQ(h_row_sums[i], expected_row_sums[i]) << "Mismatch at index " << i;
    }

    // Clean up memory
    delete[] h_y, h_row_sums;
    hipFree(d_A);
    hipFree(d_x3);
    hipFree(d_y3);
    hipFree(d_ones);
    hipFree(d_row_sums);
    hipblasDestroy(handle);
}

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
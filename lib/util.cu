#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "cublas-norm/syrk.h"
#include "util.h"

double get_time() {
   struct timeval t;
   gettimeofday(&t, NULL);
   return t.tv_sec + t.tv_usec / 1000000.0;
}

void tensor_transformation(Info *input, int filter_h, int filter_w){
    // Extract the dimensions from input->shape
    int C = input->shape[0];  // Number of channels
    int H = input->shape[1];  // Height of the input tensor
    int W = input->shape[2];  // Width of the input tensor

    // Calculate the dimensions of the output tensor
    int H_new = H - filter_h + 1;
    int W_new = W - filter_w + 1;

    // Ensure the filter dimensions are valid
    if (H_new <= 0 || W_new <= 0) {
        // Handle error: Filter size is larger than input dimensions
        return;
    }

    // Allocate memory for the reshaped tensor
    size_t output_size = H_new * W_new * filter_h * filter_w * C;
    float* reshape_tensor = (float*)malloc(output_size * sizeof(float));
    if (reshape_tensor == NULL) {
        // Handle memory allocation failure
        return;
    }

    // Perform the unfolding and reshaping
    for(int hi = 0; hi < H_new; hi++){
        for(int wi = 0; wi < W_new; wi++){
            for(int fi = 0; fi < filter_h; fi++){
                for(int fj = 0; fj < filter_w; fj++){
                    for(int c = 0; c < C; c++){
                        int h_in = hi + fi;
                        int w_in = wi + fj;
                        int index_in = c * H * W + h_in * W + w_in;
                        int index_out = hi * W_new * filter_h * filter_w * C
                                      + wi * filter_h * filter_w * C
                                      + fi * filter_w * C
                                      + fj * C
                                      + c;
                        reshape_tensor[index_out] = input->tensor[index_in];
                    }
                }
            }
        }
    }

    // Update input->tensor and input->shape
    free(input->tensor);  // Free the old tensor if it's dynamically allocated
    input->tensor = reshape_tensor;
    input->shape[0] = H_new;      // (X - filter_h + 1)
    input->shape[1] = W_new;      // (Y - filter_w + 1)
    input->shape[2] = filter_h;   // Filter height
    input->shape[3] = filter_w;   // Filter width
    input->shape[4] = C;          // Number of channels
}
// Include <torch/extension.h> and register the function only if compiling with setup.py
#ifdef BUILD_WITH_PYTORCH
#include <pybind11/numpy.h>
#include <torch/extension.h>
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("get_time", &get_time, "Get Current Time");
    pybind11::class_<Matrix>(m, "Matrix")
        .def(pybind11::init<int, double, int, double>(),
             pybind11::arg("A_N"), pybind11::arg("A_coff"),
             pybind11::arg("C_N"), pybind11::arg("C_coff"))
        .def("getA", [](const Matrix &matrix) {
            // Use the public member `A_N` directly
            return pybind11::array_t<double>({matrix.A_N, matrix.A_N}, matrix.getA());
        })
        .def("getC", [](const Matrix &matrix) {
            // Use the public member `C_N` directly
            return pybind11::array_t<double>({matrix.C_N, matrix.C_N}, matrix.getC());
        })
        .def("getAlpha", &Matrix::getAlpha)
        .def("getBeta", &Matrix::getBeta)
        .def_readonly("A_N", &Matrix::A_N)
        .def_readonly("C_N", &Matrix::C_N);

    pybind11::class_<Info>(m, "Info")
        .def(pybind11::init<int, double, int, double, int, double, double>(),
             pybind11::arg("A_N"), pybind11::arg("A_coff"),
             pybind11::arg("C_N"), pybind11::arg("C_coff"),
             pybind11::arg("iteration"), pybind11::arg("value"),
             pybind11::arg("time"))
        .def(pybind11::init<>()) // Default constructor
        .def_property_readonly("matrix", [](const Info &info) {
            return &info.matrix;
        }, pybind11::return_value_policy::reference)
        .def_readwrite("iteration", &Info::iteration)
        .def_readwrite("value", &Info::value)
        .def_readwrite("time", &Info::time);
    m.def("it_syrk", [](Info &result) {
        it_syrk(&result);
    }, "Perform iterative SYRK operation");
}

#endif